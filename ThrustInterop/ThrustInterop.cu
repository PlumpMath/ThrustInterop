#include<hip/hip_runtime_api.h>
#include<thrust/device_vector.h>
#include<thrust/fill.h>
#include<thrust/copy.h>
#include<thrust/inner_product.h>

typedef thrust::device_vector<double> Vector;

extern "C"
{
	__declspec(dllexport) int _stdcall GetDeviceCount()
	{
		int count;
		::hipGetDeviceCount(&count);

		return count;
	}

	__declspec(dllexport) Vector* _stdcall Create(int size, int deviceID)
	{
		::hipSetDevice(deviceID);

		Vector* vec = new Vector(size);

		return vec;
	}
	
	__declspec(dllexport) void _stdcall Fill(Vector* vec, int size, double value, int deviceID)
	{
		::hipSetDevice(deviceID);
		thrust::fill_n(vec->begin(), size, value);
	}
	
	__declspec(dllexport) double _stdcall Length2(Vector* vec, int size, int deviceID)
	{
		::hipSetDevice(deviceID);
		return thrust::inner_product(vec->begin(),vec->begin() + size, vec->begin(), 0.0);
	}

	__declspec(dllexport) void _stdcall CopyTo(Vector* source, double destination[], int size, int deviceID)
	{
		::hipSetDevice(deviceID);
		thrust::copy_n(source->begin(), size, destination);
	}
	
	__declspec(dllexport) void _stdcall Delete(Vector* vec, int deviceID)
	{
		::hipSetDevice(deviceID);
		delete vec;
	}
}